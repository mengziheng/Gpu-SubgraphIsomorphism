#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <sstream>
#include <string>
#include <hipcub/hipcub.hpp>
#include <math.h>
#include <hip/hip_cooperative_groups.h>
#include <thrust/extrema.h>
#include <thrust/device_ptr.h>
#include <stdio.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <sys/time.h>
#include <unistd.h>

// 添加了最后一层的时候不写回
// 添加了不用做交集的时候不用从neighbour list读入到candidate
// 添加了用做交集的时候不用从neighbour list读入到candidate

#define FULL_MASK 0xffffffff

using namespace cooperative_groups;

using namespace std;

int vertex_count = 0, vCount = 0, uMax = 0, edge_count;
float load_factor = 0.25;
int load_factor_inverse = 1 / load_factor;
int bucket_size = 4;
int bucket_num;
int parameter = load_factor_inverse / bucket_size;
int max_degree;                // 记录最大度数
int pattern_vertex_number = 3; // pattern的节点数量

inline off_t fsize(const char *filename)
{
    struct stat st;
    if (stat(filename, &st) == 0)
    {
        return st.st_size;
    }
    return -1;
}

double wtime()
{
    double time[2];
    struct timeval time1;
    gettimeofday(&time1, NULL);

    time[0] = time1.tv_sec;
    time[1] = time1.tv_usec;

    return time[0] + time[1] * 1.0e-6;
}

double getDeltaTime(double &startTime)
{
    double deltaTime = wtime() - startTime;
    startTime = wtime();
    return deltaTime;
}

static void HandError(hipError_t err, const char *file, int line)
{
    if (err != hipSuccess)
    {
        printf("\n%s in %s at line %d\n",
               hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}
#define HRR(err) (HandError(err, __FILE__, __LINE__))

__global__ void buildHashTableOffset(int *hash_tables_offset, int *csr_row_offset, int vertex_count, int parameter)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = tid; i < vertex_count + 1; i += stride)
        hash_tables_offset[i] = csr_row_offset[i] * parameter;
}

__global__ void buildHashTable(int *hash_tables_offset, int *hash_tables, int *csr_column_index, int *vertex_list, int edge_count, int bucket_size, int load_factor_inverse, int bucket_num)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int key;
    int vertex;
    int hash_table_start;
    int hash_table_end;
    int hash_table_length;
    int value;
    for (int i = tid; i < edge_count; i += stride)
    {
        key = csr_column_index[i];
        vertex = vertex_list[i];
        hash_table_start = hash_tables_offset[vertex];
        hash_table_end = hash_tables_offset[vertex + 1];
        hash_table_length = hash_table_end - hash_table_start;
        // hash function = % k
        value = key % hash_table_length;

        int index = 0;
        while (atomicCAS(&hash_tables[hash_table_start + value + index * bucket_num], -1, key) != -1)
        {
            index++;
            if (index == bucket_size)
            {
                index = 0;
                value++;
                if (value == hash_table_length)
                    value = 0;
            }
        }
    }
}

__inline__ __device__ void swap(int &a, int &b)
{
    int t = a;
    a = b;
    b = t;
}

__inline__ __device__ bool search_in_hashtable(int x, int bucket_num, int bucket_size, int hash_table_len, int *hash_table)
{
    int value = x % hash_table_len;
    int *cmp = hash_table + value;
    int index = 0;
    while (*cmp != -1)
    {
        if (*cmp == x)
        {
            return true;
        }
        cmp = cmp + bucket_num;
        index++;
        if (index == bucket_size)
        {
            value++;
            index = 0;
            if (value == hash_table_len)
                value = 0;
            cmp = &hash_table[value];
        }
    }
    return false;
}

// candidates_of_all_warp
// my_candidates_for_all_mapping
// my_candidates

// 还没有解决如何动态的传入数组
// h : height of subtree; h = pattern vertex number
__global__ void DFSKernel(int vertex_count, int bucket_num, int max_degree, int h, int bucket_size, int *intersection_orders, int *intersection_offset, int *csr_row_offset, int *csr_row_value, int *csr_column_index, int *hash_tables_offset, int *hash_tables, int *candidates_of_all_warp, int *sum)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // threadid
    int wid = tid / 32;                              // warpid
    int lid = tid % 32;                              // landid
    int stride = blockDim.x * gridDim.x / 32;
    int candidate_number_array[3]; // 记录一下每一层保存的数据大小
    int next_candidate_array[3];   // 记录一下每一层保存的数据大小
    int mapping[3];                // 记录每次的中间结果
    int *my_candidates_for_all_mapping = candidates_of_all_warp + (long long)wid * h * max_degree;
    int my_count = 0;
    // each warp process a subtree (an probe item)
    for (int start_vertex = wid; start_vertex < vertex_count; start_vertex += stride)
    {
        mapping[0] = start_vertex;
        int level = 0;
        for (;;)
        {
            level++;
            int &candidate_number = candidate_number_array[level];
            next_candidate_array[level] = -1;
            candidate_number = 0;
            __syncwarp();
            // find possible connection and maintain in S
            int intersection_order_start = intersection_offset[level - 1];
            int intersection_order_length = intersection_offset[level] - intersection_order_start;
            int mapped_intersection_order[2]; // wzb: refine it
            for (int i = 0; i < intersection_order_length; i++)
            {
                mapped_intersection_order[i] = mapping[intersection_orders[intersection_order_start + i]];
            }

            // get degree
            int neighbour_numbers[2];
            neighbour_numbers[0] = csr_row_value[mapped_intersection_order[0]];
            for (int i = 1; i < intersection_order_length; i++)
            {
                neighbour_numbers[i] = csr_row_value[mapped_intersection_order[i]];
                if (neighbour_numbers[i] < neighbour_numbers[0])
                {
                    swap(mapped_intersection_order[i], mapped_intersection_order[0]);
                    swap(neighbour_numbers[i], neighbour_numbers[0]);
                }
            }

            // Start intersection, load neighbor of m1 into my_candidates
            int *my_candidates = my_candidates_for_all_mapping + level * max_degree;
            int cur_vertex = mapped_intersection_order[0];
            int *cur_neighbor_list = csr_column_index + csr_row_offset[cur_vertex];

            // 要做交集，则不抄出来,记录一下这个neighbour list所在的位置
            if (intersection_order_length == 1 && lid == 0)
            {
                my_candidates[0] = -1;
                my_candidates[1] = (int)((long)cur_neighbor_list >> 32);
                my_candidates[2] = (int)((long)cur_neighbor_list & 0xFFFFFFFF);
            }

            // intersect
            candidate_number = neighbour_numbers[0];
            for (int j = 1; j < intersection_order_length; j++)
            {
                cur_vertex = mapped_intersection_order[j];
                int *cur_hashtable = hash_tables + hash_tables_offset[cur_vertex];
                int len = hash_tables_offset[cur_vertex + 1] - hash_tables_offset[cur_vertex]; // len记录当前hash_table的长度

                int candidate_number_previous = candidate_number;
                candidate_number = 0;
                for (int i = lid; i < candidate_number_previous; i += 32)
                {
                    int item;
                    if (j == 1)
                        item = cur_neighbor_list[i];
                    else
                        item = my_candidates[i];
                    int is_exist = search_in_hashtable(item, bucket_num, bucket_size, len, cur_hashtable);
                    int count = __reduce_add_sync(__activemask(), is_exist);
                    if (is_exist)
                    {
                        coalesced_group active = coalesced_threads();
                        // 最后一层不写回
                        if (level != h - 1)
                            my_candidates[active.thread_rank() + candidate_number] = item;
                    }
                    candidate_number += count;
                }
                candidate_number = __shfl_sync(FULL_MASK, candidate_number, 0);
            }
            __syncwarp();

            if (level == h - 1)
            {
                if (lid == 0)
                {
                    my_count += candidate_number;
                }
                __syncwarp();
                level--;
            }
            for (;; level--)
            {
                if (level == 0)
                    break;
                next_candidate_array[level]++;
                if (next_candidate_array[level] < candidate_number_array[level])
                {
                    // 如果是-1，说明没有写入，直接从neighbour list读出的
                    if (my_candidates_for_all_mapping[level * max_degree] == -1)
                    {
                        int *result;
                        result = (int *)(((long)my_candidates_for_all_mapping[level * max_degree + 1] << 32) | ((long)my_candidates_for_all_mapping[level * max_degree + 2] & 0xFFFFFFFF));
                        mapping[level] = result[next_candidate_array[level]];
                    }
                    else
                        mapping[level] = my_candidates_for_all_mapping[level * max_degree + next_candidate_array[level]];
                    break;
                }
            }
            if (level == 0)
                break;
        }
    }
    if (lid == 0)
    {
        atomicAdd(sum, my_count);
    }
}

int main(int argc, char *argv[])
{
    int *csr_column_index;
    int *csr_row_offset;
    int *csr_row_value;
    int *vertex_list;

    string s_begin = "../version_2/begin.bin";  // 记录索引，而且已经维护了最后一位
    string s_adj = "../version_2/adjacent.bin"; // 记录邻居节点
    string s_degree = "../version_2/edge";      // 记录邻居节点数目
    string s_vertex = "../version_2/vertex";

    char *begin_file = const_cast<char *>(s_begin.c_str());
    char *adj_file = const_cast<char *>(s_adj.c_str());
    char *degree_file = const_cast<char *>(s_degree.c_str());
    char *vertex_file = const_cast<char *>(s_vertex.c_str());

    int vertex_count = fsize(begin_file) / sizeof(int) - 1;
    int edge_count = fsize(adj_file) / sizeof(int);

    FILE *pFile1 = fopen(begin_file, "rb");
    csr_row_offset = (int *)malloc(fsize(begin_file));
    fread(csr_row_offset, sizeof(int), vertex_count + 1, pFile1);
    fclose(pFile1);

    FILE *pFile2 = fopen(adj_file, "rb");
    csr_column_index = (int *)malloc(fsize(adj_file));
    fread(csr_column_index, sizeof(int), edge_count, pFile2);
    fclose(pFile2);

    FILE *pFile3 = fopen(degree_file, "rb");
    csr_row_value = (int *)malloc(fsize(degree_file));
    fread(csr_row_value, sizeof(int), vertex_count, pFile3);
    fclose(pFile3);

    FILE *pFile4 = fopen(vertex_file, "rb");
    vertex_list = (int *)malloc(fsize(vertex_file));
    fread(vertex_list, sizeof(int), edge_count, pFile4);
    fclose(pFile4);

    int *d_csr_row_offset;
    int *d_csr_column_index, *d_csr_row_value, *d_vertex_list;
    HRR(hipMalloc((void **)&d_csr_row_offset, sizeof(int) * (vertex_count + 1)));
    HRR(hipMalloc((void **)&d_csr_column_index, sizeof(int) * (edge_count)));
    HRR(hipMalloc((void **)&d_vertex_list, sizeof(int) * (edge_count)));
    HRR(hipMalloc((void **)&d_csr_row_value, sizeof(int) * (vertex_count + 1)));

    HRR(hipMemcpy(d_csr_row_value, csr_row_value, sizeof(int) * (vertex_count + 1), hipMemcpyHostToDevice));
    HRR(hipMemcpy(d_csr_row_offset, csr_row_offset, sizeof(int) * (vertex_count + 1), hipMemcpyHostToDevice));
    HRR(hipMemcpy(d_csr_column_index, csr_column_index, sizeof(int) * edge_count, hipMemcpyHostToDevice));
    HRR(hipMemcpy(d_vertex_list, vertex_list, sizeof(int) * edge_count, hipMemcpyHostToDevice));

    bucket_num = edge_count * load_factor_inverse / bucket_size;
    cout << "graph vertex number is : " << vertex_count << endl;
    cout << "graph edge number is : " << edge_count << endl;
    cout << "graph load_factor_inverse is : " << load_factor_inverse << endl;
    cout << "graph bucket_size is : " << bucket_size << endl;
    cout << "graph parameter is : " << parameter << endl;
    cout << "graph bucket_num is : " << bucket_num << endl;

    // 查看下可用share memory的最大值
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0); // 假设设备号为0
    size_t sharedMemPerBlock = deviceProp.sharedMemPerBlock;
    cout << "share memory size : " << sharedMemPerBlock << endl;

    // compute max degree
    int *d_max_degree;
    hipMalloc(&d_max_degree, sizeof(int));
    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_csr_row_value, d_max_degree, vertex_count);
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_csr_row_value, d_max_degree, vertex_count);
    HRR(hipMemcpy(&max_degree, d_max_degree, sizeof(int), hipMemcpyDeviceToHost));
    std::cout << "max degree is: " << max_degree << std::endl;

    // build hash table in device
    int *d_hash_tables;
    HRR(hipMalloc(&d_hash_tables, (long long)bucket_size * bucket_num * sizeof(int)));
    HRR(hipMemset(d_hash_tables, -1, (long long)bucket_size * bucket_num * sizeof(int)));
    int *d_hash_tables_offset;
    HRR(hipMalloc(&d_hash_tables_offset, (vertex_count + 1) * sizeof(int)));
    buildHashTableOffset<<<216, 1024>>>(d_hash_tables_offset, d_csr_row_offset, vertex_count, parameter);
    buildHashTable<<<216, 1024>>>(d_hash_tables_offset, d_hash_tables, d_csr_column_index, d_vertex_list, edge_count, bucket_size, load_factor_inverse, bucket_num);

    // DFS
    int *d_ir; // intermediate result;
    // refine the malloc
    HRR(hipMalloc(&d_ir, (long long)216 * 32 * max_degree * pattern_vertex_number * sizeof(int)));
    cout << "ir memory size is : " << 216 * 32 * max_degree * pattern_vertex_number << endl;
    // 先提前假定一下三角形的顺序
    int intersection_orders[4] = {0, 0, 1};
    int intersection_offset[4] = {0, 1, 3};
    int *d_intersection_orders;
    int intersection_size = 4;
    HRR(hipMalloc(&d_intersection_orders, intersection_size * sizeof(int)));
    HRR(hipMemcpy(d_intersection_orders, intersection_orders, intersection_size * sizeof(int), hipMemcpyHostToDevice));
    int *d_intersection_offset;
    HRR(hipMalloc(&d_intersection_offset, intersection_size * sizeof(int)));
    HRR(hipMemcpy(d_intersection_offset, intersection_offset, intersection_size * sizeof(int), hipMemcpyHostToDevice));
    int h = 3;
    int *d_sum;
    HRR(hipMalloc(&d_sum, 4));
    HRR(hipMemset(d_sum, 0, 4));
    // double start_time = wtime();

    double t1 = wtime();
    double cmp_time;
    double time_start;
    double max_time = 0;
    double min_time = 1000;
    double ave_time = 0;
    for (int i = 0; i < 1000; i++)
    {
        HRR(hipMemset(d_sum, 0, 4));
        time_start = clock();
        DFSKernel<<<216, 1024>>>(vertex_count, bucket_num, max_degree, h, bucket_size, d_intersection_orders, d_intersection_offset, d_csr_row_offset, d_csr_row_value, d_csr_column_index, d_hash_tables_offset, d_hash_tables, d_ir, d_sum);
        HRR(hipDeviceSynchronize());
        cmp_time = clock() - time_start;
        cmp_time = cmp_time / CLOCKS_PER_SEC;
        if (cmp_time > max_time)
            max_time = cmp_time;
        if (cmp_time < min_time)
            min_time = cmp_time;
        ave_time += cmp_time;
        HRR(hipFree(d_ir));
        HRR(hipMalloc(&d_ir, (long long)216 * 32 * max_degree * pattern_vertex_number * sizeof(int)));
    }
    std::cout << "max time: " << max_time * 1000 << " ms" << std::endl;
    std::cout << "min time: " << min_time * 1000 << " ms" << std::endl;
    std::cout << "average time: " << ave_time << " ms" << std::endl;

    int sum;
    hipMemcpy(&sum, d_sum, 4, hipMemcpyDeviceToHost);
    printf("triangle count is %d\n", sum);
}