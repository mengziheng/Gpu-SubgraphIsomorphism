#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <sstream>
#include <string>
#include <hipcub/hipcub.hpp>
#include <math.h>
#include <hip/hip_cooperative_groups.h>
#include <thrust/extrema.h>
#include <thrust/device_ptr.h>
#include <stdio.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <sys/time.h>
#include <unistd.h>
#include "main.cuh"

// 添加了location
// 添加了写回到candidate时候，先写到share memory，再写到global memory
// 108 * 1024 -> 216 * 1024

#define FULL_MASK 0xffffffff
#define MIN_BUCKET_NUM 8
#define CHUNK_SIZE 1
#define BUCKET_SIZE 4
#define map_value(hash_value, bucket_num) (((hash_value) % MIN_BUCKET_NUM) * (bucket_num / MIN_BUCKET_NUM) + (hash_value) / MIN_BUCKET_NUM)
#define H 4

using namespace cooperative_groups;

using namespace std;

int vertex_count = 0, vCount = 0, uMax = 0, edge_count;
float load_factor = 0.25;
int load_factor_inverse = 1 / load_factor;
long long bucket_num;
int parameter = load_factor_inverse / BUCKET_SIZE;
int max_degree; // 记录最大度数

inline off_t fsize(const char *filename)
{
    struct stat st;
    if (stat(filename, &st) == 0)
    {
        return st.st_size;
    }
    return -1;
}

double wtime()
{
    double time[2];
    struct timeval time1;
    gettimeofday(&time1, NULL);

    time[0] = time1.tv_sec;
    time[1] = time1.tv_usec;

    return time[0] + time[1] * 1.0e-6;
}

double getDeltaTime(double &startTime)
{
    double deltaTime = wtime() - startTime;
    startTime = wtime();
    return deltaTime;
}

static void HandError(hipError_t err, const char *file, int line)
{
    if (err != hipSuccess)
    {
        printf("\n%s in %s at line %d\n",
               hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}
#define HRR(err) (HandError(err, __FILE__, __LINE__))

__global__ void buildHashTablelens(int *hash_tables_lens, int *csr_row_value, int vertex_count, int parameter)
{

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int len;
    int log;
    for (int i = tid; i < vertex_count; i += stride)
    {
        len = csr_row_value[i] * parameter;
        log = log2f(len);
        len = powf(2, log) == len ? len : powf(2, log + 1);
        if (len < 8 && len != 0)
            len = 8;
        hash_tables_lens[i] = len;
    }
}

__global__ void buildHashTable(long long *hash_tables_offset, int *hash_tables, int *csr_column_index, int *vertex_list, int edge_count, int load_factor_inverse, long long bucket_num)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int key;
    int vertex;
    long long hash_table_start;
    long long hash_table_end;
    int hash_table_length;
    int value;
    int mapped_value;
    for (int i = tid; i < edge_count; i += stride)
    {
        key = csr_column_index[i];
        vertex = vertex_list[i];
        hash_table_start = hash_tables_offset[vertex];
        hash_table_end = hash_tables_offset[vertex + 1];
        hash_table_length = hash_table_end - hash_table_start;
        // hash function = % k
        value = key % hash_table_length;
        mapped_value = map_value(value, hash_table_length);
        int index = 0;
        while (atomicCAS(&hash_tables[hash_table_start + mapped_value + bucket_num * index], -1, key) != -1)
        {
            index++;
            if (index == BUCKET_SIZE)
            {
                index = 0;
                mapped_value++;
                if (mapped_value == hash_table_length)
                    mapped_value = 0;
            }
        }
    }
}

__inline__ __device__ void swap(int &a, int &b)
{
    int t = a;
    a = b;
    b = t;
}

__inline__ __device__ bool search_in_hashtable(int x, long long bucket_num, int hash_table_len, int *hash_table)
{
    int mapped_value = map_value(x % hash_table_len, hash_table_len);
    int *cmp = hash_table + mapped_value;
    int index = 0;
    while (*cmp != -1)
    {
        if (*cmp == x)
        {
            return true;
        }
        cmp = cmp + bucket_num;
        index++;
        if (index == BUCKET_SIZE)
        {
            mapped_value++;
            index = 0;
            if (mapped_value == hash_table_len)
                mapped_value = 0;
            cmp = &hash_table[mapped_value];
        }
    }
    return false;
}

// candidates_of_all_warp
// my_candidates_for_all_mapping
// my_candidates

// 还没有解决如何动态的传入数组
// h : height of subtree; h = pattern vertex number
__global__ void DFSKernel(int vertex_count, long long bucket_num, int max_degree, int *intersection_orders, int *intersection_offset, int *restriction, int *csr_row_offset, int *csr_row_value, int *csr_column_index, long long *hash_tables_offset, int *hash_tables, int *candidates_of_all_warp, unsigned long long *sum, int *G_INDEX)
{
    int wid = (blockIdx.x * blockDim.x + threadIdx.x) / 32; // warpid
    int lid = threadIdx.x % 32;                             // landid
    int next_candidate_array[H];                            // 记录一下每一层保存的数据大小
    int mapping[H];                                         // 记录每次的中间结果
    int *my_candidates_for_all_mapping = candidates_of_all_warp + (long long)wid * H * max_degree;
    unsigned long long my_count = 0;
    // each warp process a subtree (an probe item)
    int start_vertex = wid * CHUNK_SIZE;
    int vertex_end = start_vertex + CHUNK_SIZE;
    int level;
    // each warp process a subtree (an probe item)
    while (start_vertex < vertex_count)
    {
        mapping[0] = start_vertex;
        level = 0;
        for (;;)
        {
            level++;
            int candidate_number;
            candidate_number = 0;
            // find possible connection and maintain in S
            int intersection_order_start = intersection_offset[level - 1];
            int intersection_order_length = intersection_offset[level] - intersection_order_start;
            // get degree
            int min_neighbour_numbers;
            int min_neighbour_vertex = mapping[intersection_orders[intersection_order_start]];
            min_neighbour_numbers = csr_row_value[min_neighbour_vertex];
            for (int i = 1; i < intersection_order_length; i++)
            {
                if (csr_row_value[mapping[intersection_orders[intersection_order_start + i]]] < min_neighbour_numbers)
                {
                    min_neighbour_vertex = mapping[intersection_orders[intersection_order_start + i]];
                    min_neighbour_numbers = csr_row_value[min_neighbour_vertex];
                }
            }
            // Start intersection, load neighbor of m1 into my_candidates
            int *my_candidates = my_candidates_for_all_mapping + level * max_degree;
            int cur_vertex;
            int *cur_neighbor_list = csr_column_index + csr_row_offset[min_neighbour_vertex];

            if (intersection_order_length == 1)
            {
                int count = 0;
                for (int i = lid; i < min_neighbour_numbers; i += 32)
                {
                    int flag = restriction[level] != -1 && cur_neighbor_list[i] > mapping[restriction[level]];
                    if (flag)
                    {
                        coalesced_group active = coalesced_threads();
                        my_candidates[active.thread_rank() + candidate_number] = cur_neighbor_list[i];
                        count = active.num_threads();
                        // ir[active.thread_rank() + ptr[in_block_wid] + wid * max_degree + level * 1024 * 216 / 32 * max_degree] = thread_cache[i];
                    }
                    candidate_number += __reduce_add_sync(__activemask(), flag);
                }
            }
            else
            {
                candidate_number = min_neighbour_numbers;
            }
            __syncwarp();
            // if (wid == 0 && lid == 0)
            //     printf("level : %d candidate_number : %d\n", candidate_number);
            for (int j = 0; j < intersection_order_length; j++)
            {
                cur_vertex = mapping[intersection_orders[intersection_order_start + j]];
                if (cur_vertex == min_neighbour_vertex)
                    continue;
                int *cur_hashtable = hash_tables + hash_tables_offset[cur_vertex];
                int len = int(hash_tables_offset[cur_vertex + 1] - hash_tables_offset[cur_vertex]); // len记录当前hash_table的长度

                int candidate_number_previous = candidate_number;
                candidate_number = 0;
                for (int i = lid; i < candidate_number_previous; i += 32)
                {
                    int item;
                    if (j == 0 || j == 1 && mapping[intersection_orders[intersection_order_start]] == min_neighbour_vertex)
                        item = cur_neighbor_list[i];
                    else
                        item = my_candidates[i];
                    int cmp = mapping[restriction[level]];
                    int is_exist;
                    if (restriction[level] != -1 && item <= cmp)
                        is_exist = false;
                    else
                        is_exist = search_in_hashtable(item, bucket_num, len, cur_hashtable);
                    __syncwarp();
                    if (level != H - 1 && is_exist)
                    {
                        coalesced_group active = coalesced_threads();
                        // 最后一层不写回
                        my_candidates[active.thread_rank() + candidate_number] = item;
                    }
                    candidate_number += __reduce_add_sync(__activemask(), is_exist);
                }
                // 这一行是否可以删掉？？？
                candidate_number = __shfl_sync(FULL_MASK, candidate_number, 0);
            }
            next_candidate_array[level] = candidate_number;
            // if (wid == 0 && lid == 0)
            // printf("level : %d mapping : %d %d %d %d\n", level, mapping[0], mapping[1], mapping[2], mapping[3]);
            if (level == H - 1)
            {
                if (lid == 0)
                {
                    my_count += candidate_number;
                }
                level--;
            }
            for (;; level--)
            {
                if (level == 0)
                    break;
                next_candidate_array[level]--;
                if (next_candidate_array[level] > -1)
                {
                    mapping[level] = my_candidates_for_all_mapping[level * max_degree + next_candidate_array[level]];
                    break;
                }
            }
            if (level == 0)
                break;
        }
        start_vertex++;
        if (start_vertex == vertex_end)
        {
            if (lid == 0)
            {
                start_vertex = atomicAdd(&G_INDEX[0], CHUNK_SIZE);
            }
            start_vertex = __shfl_sync(0xffffffff, start_vertex, 0);
            vertex_end = start_vertex + CHUNK_SIZE;
        }
    }
    if (wid < vertex_count && lid == 0)
        printf("tid : %d count for vertex %d is %d\n", blockIdx.x * blockDim.x + threadIdx.x, wid, my_count);
    if (lid == 0)
    {
        atomicAdd(sum, my_count);
    }
}

int main(int argc, char *argv[])
{
    int *csr_column_index;
    int *csr_row_offset;
    int *csr_row_value;
    int *vertex_list;

    string Infilename = "4-cycle";
    string pattern = "4-cycle";
    if (argc > 1)
    {
        Infilename = argv[1];
        pattern = argv[2];
    }

    // string s_begin = "/data/zh_dataset/dataforclique/" + Infilename + "/begin.bin";  // 记录索引，而且已经维护了最后一位
    // string s_adj = "/data/zh_dataset/dataforclique/" + Infilename + "/adjacent.bin"; // 记录邻居节点
    // string s_degree = "/data/zh_dataset/dataforclique/" + Infilename + "/edge";      // 记录邻居节点数目
    // string s_vertex = "/data/zh_dataset/dataforclique/" + Infilename + "/vertex";

    string s_begin = "/data/zh_dataset/dataforgeneral/" + Infilename + "/begin.bin"; // 记录索引，而且已经维护了最后一位
    string s_adj = "/data/zh_dataset/dataforgeneral/" + Infilename + "/adj.bin";     // 记录邻居节点
    string s_degree = "/data/zh_dataset/dataforgeneral/" + Infilename + "/edge";     // 记录邻居节点数目
    string s_vertex = "/data/zh_dataset/dataforgeneral/" + Infilename + "/vertex";

    char *begin_file = const_cast<char *>(s_begin.c_str());
    char *adj_file = const_cast<char *>(s_adj.c_str());
    char *degree_file = const_cast<char *>(s_degree.c_str());
    char *vertex_file = const_cast<char *>(s_vertex.c_str());

    int vertex_count = fsize(begin_file) / sizeof(int) - 1;
    int edge_count = fsize(adj_file) / sizeof(int);

    FILE *pFile1 = fopen(begin_file, "rb");
    if (!pFile1)
    {
        cout << "error" << endl;
        // return 0;
    }
    csr_row_offset = (int *)malloc(fsize(begin_file));
    fread(csr_row_offset, sizeof(int), vertex_count + 1, pFile1);
    fclose(pFile1);

    FILE *pFile2 = fopen(adj_file, "rb");
    if (!pFile2)
    {
        cout << "error" << endl;
        // return 0;
    }
    csr_column_index = (int *)malloc(fsize(adj_file));
    fread(csr_column_index, sizeof(int), edge_count, pFile2);
    fclose(pFile2);

    // int j = 0;
    // int z = 0;
    // for (int i = 0; i < edge_count; i++)
    // {
    //     printf("%d ", csr_column_index[i]);
    //     j++;
    //     if (j == csr_row_offset[z + 1])
    //     {
    //         printf("is neighbour of %d\n", z);
    //         j == 0;
    //         z++;
    //     }
    // }
    // printf("\n");

    FILE *pFile3 = fopen(degree_file, "rb");
    if (!pFile3)
    {
        cout << "error" << endl;
        // return 0;
    }
    csr_row_value = (int *)malloc(fsize(degree_file));
    fread(csr_row_value, sizeof(int), vertex_count, pFile3);
    fclose(pFile3);

    FILE *pFile4 = fopen(vertex_file, "rb");
    if (!pFile4)
    {
        cout << "error" << endl;
        // return 0;
    }
    vertex_list = (int *)malloc(fsize(vertex_file));
    fread(vertex_list, sizeof(int), edge_count, pFile4);
    fclose(pFile4);

    int *d_csr_row_offset;
    int *d_csr_column_index, *d_csr_row_value, *d_vertex_list;
    HRR(hipMalloc((void **)&d_csr_row_offset, sizeof(int) * (vertex_count + 1)));
    HRR(hipMalloc((void **)&d_csr_column_index, sizeof(int) * (edge_count)));
    HRR(hipMalloc((void **)&d_vertex_list, sizeof(int) * (edge_count)));
    HRR(hipMalloc((void **)&d_csr_row_value, sizeof(int) * (vertex_count + 1)));

    HRR(hipMemcpy(d_csr_row_value, csr_row_value, sizeof(int) * (vertex_count + 1), hipMemcpyHostToDevice));
    HRR(hipMemcpy(d_csr_row_offset, csr_row_offset, sizeof(int) * (vertex_count + 1), hipMemcpyHostToDevice));
    HRR(hipMemcpy(d_csr_column_index, csr_column_index, sizeof(int) * edge_count, hipMemcpyHostToDevice));
    HRR(hipMemcpy(d_vertex_list, vertex_list, sizeof(int) * edge_count, hipMemcpyHostToDevice));

    cout << "graph vertex number is : " << vertex_count << endl;
    cout << "graph edge number is : " << edge_count << endl;
    cout << "graph load_factor_inverse is : " << load_factor_inverse << endl;
    cout << "graph BUCKET_SIZE is : " << BUCKET_SIZE << endl;

    // 查看下可用share memory的最大值
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0); // 假设设备号为0
    size_t sharedMemPerBlock = deviceProp.sharedMemPerBlock;
    cout << "share memory size per block : " << sharedMemPerBlock << endl;
    cout << "registers number per block : " << deviceProp.regsPerBlock << endl;

    // compute max degree
    int *d_max_degree;
    hipMalloc(&d_max_degree, sizeof(int));
    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_csr_row_value, d_max_degree, vertex_count);
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_csr_row_value, d_max_degree, vertex_count);
    HRR(hipMemcpy(&max_degree, d_max_degree, sizeof(int), hipMemcpyDeviceToHost));
    std::cout << "max degree is: " << max_degree << std::endl;

    // get bucket_num
    int *d_hash_tables_lens;
    HRR(hipMalloc(&d_hash_tables_lens, vertex_count * sizeof(int)));
    buildHashTablelens<<<216, 1024>>>(d_hash_tables_lens, d_csr_row_value, vertex_count, parameter);
    // exclusiveSum
    long long *d_hash_tables_offset;
    HRR(hipMalloc(&d_hash_tables_offset, (vertex_count + 1) * sizeof(long long)));
    d_temp_storage = NULL;
    temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_hash_tables_lens, d_hash_tables_offset, vertex_count);
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_hash_tables_lens, d_hash_tables_offset, vertex_count);

    // get sum
    int last_count;
    long long last_sum;
    HRR(hipMemcpy(&last_count, d_hash_tables_lens + vertex_count - 1, sizeof(int), hipMemcpyDeviceToHost));
    HRR(hipMemcpy(&last_sum, d_hash_tables_offset + vertex_count - 1, sizeof(long long), hipMemcpyDeviceToHost));
    bucket_num = last_sum + last_count;
    HRR(hipMemcpy(d_hash_tables_offset + vertex_count, &bucket_num, sizeof(long long), hipMemcpyHostToDevice));

    // build hash table in device
    int *d_hash_tables;
    HRR(hipMalloc(&d_hash_tables, BUCKET_SIZE * bucket_num * sizeof(int)));
    HRR(hipMemset(d_hash_tables, -1, BUCKET_SIZE * bucket_num * sizeof(int)));
    cout << "bucket_num is : " << bucket_num << endl;
    cout << "hash table size is : " << BUCKET_SIZE * bucket_num * sizeof(int) << endl;

    buildHashTable<<<216, 1024>>>(d_hash_tables_offset, d_hash_tables, d_csr_column_index, d_vertex_list, edge_count, load_factor_inverse, bucket_num);

    // DFS
    int *d_ir; // intermediate result;
    // refine the malloc
    HRR(hipMalloc(&d_ir, (long long)216 * 32 * max_degree * H * sizeof(int)));
    cout << "ir memory size is : " << 216 * 32 * max_degree * H / 1024 / 1024 << "MB" << endl;
    int intersection_orders[10];
    int intersection_offset[10];
    int restriction[10];

    int intersection_size;
    // 先提前假定一下三角形的顺序
    if (pattern.compare("triangle") == 0)
    {
        int tmp_intersection_orders[4] = {0, 0, 1};
        int tmp_intersection_offset[4] = {0, 1, 3};
        intersection_size = 4;
        memcpy(intersection_orders, tmp_intersection_orders, intersection_size * sizeof(int));
        memcpy(intersection_offset, tmp_intersection_offset, intersection_size * sizeof(int));
    }
    else if (pattern.compare("4-cycle") == 0)
    {
        int tmp_intersection_orders[5] = {0, 0, 1, 2};
        int tmp_intersection_offset[5] = {0, 1, 2, 4};
        int tmp_restriction[5] = {-1, 0, 1, 0};
        intersection_size = 5;
        memcpy(intersection_orders, tmp_intersection_orders, intersection_size * sizeof(int));
        memcpy(intersection_offset, tmp_intersection_offset, intersection_size * sizeof(int));
        memcpy(restriction, tmp_restriction, 5 * sizeof(int));
    }
    int *d_intersection_orders;
    HRR(hipMalloc(&d_intersection_orders, intersection_size * sizeof(int)));
    HRR(hipMemcpy(d_intersection_orders, intersection_orders, intersection_size * sizeof(int), hipMemcpyHostToDevice));
    int *d_intersection_offset;
    HRR(hipMalloc(&d_intersection_offset, intersection_size * sizeof(int)));
    HRR(hipMemcpy(d_intersection_offset, intersection_offset, intersection_size * sizeof(int), hipMemcpyHostToDevice));
    int *d_restriction;
    HRR(hipMalloc(&d_restriction, 5 * sizeof(int)));
    HRR(hipMemcpy(d_restriction, restriction, 5 * sizeof(int), hipMemcpyHostToDevice));
    printf("%d %d %d %d\n", restriction[0], restriction[1], restriction[2], restriction[3]);
    int *G_INDEX;
    int block_num = 216;
    int block_size = 512;
    int temp = block_num * block_size / 32 * CHUNK_SIZE;
    HRR(hipMalloc(&G_INDEX, sizeof(int)));
    HRR(hipMemcpy(G_INDEX, &temp, sizeof(int), hipMemcpyHostToDevice));

    unsigned long long *d_sum;
    HRR(hipMalloc(&d_sum, sizeof(unsigned long long)));
    HRR(hipMemset(d_sum, 0, sizeof(unsigned long long)));
    // double start_time = wtime();

    int numBlocks; // Occupancy in terms of active blocks
    int blockSize = 1024;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocks, DFSKernel, blockSize, 0);
    printf("numBlocks : %d\n", numBlocks);

    double cmp_time;
    double time_start;
    double max_time = 0;
    double min_time = 1000;
    double ave_time = 0;
    for (int i = 0; i < 3; i++)
    {
        HRR(hipMemset(d_sum, 0, sizeof(unsigned long long)));
        time_start = clock();
        DFSKernel<<<block_num, block_size>>>(vertex_count, bucket_num, max_degree, d_intersection_orders, d_intersection_offset, d_restriction, d_csr_row_offset, d_csr_row_value, d_csr_column_index, d_hash_tables_offset, d_hash_tables, d_ir, d_sum, G_INDEX);
        HRR(hipDeviceSynchronize());
        cmp_time = clock() - time_start;
        cmp_time = cmp_time / CLOCKS_PER_SEC;
        if (cmp_time > max_time)
            max_time = cmp_time;
        if (cmp_time < min_time)
            min_time = cmp_time;
        ave_time += cmp_time;
        HRR(hipFree(d_ir));
        HRR(hipMalloc(&d_ir, (long long)216 * 32 * max_degree * H * sizeof(int)));
        HRR(hipMemcpy(G_INDEX, &temp, sizeof(int), hipMemcpyHostToDevice));
    }
    std::cout << "max time: " << max_time * 1000 << " ms" << std::endl;
    std::cout << "min time: " << min_time * 1000 << " ms" << std::endl;
    std::cout << "average time: " << ave_time / 10 * 1000 << " ms" << std::endl;

    long long sum;
    hipMemcpy(&sum, d_sum, sizeof(long long), hipMemcpyDeviceToHost);
    printf("triangle count is %lld\n", sum);
}